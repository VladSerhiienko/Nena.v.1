#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <sal.h>

#include <random>

// suppress warning:
// nonstandard extension used 
#pragma warning(disable: 4482)

namespace nena { namespace cuda
{
	typedef hipError_t error;
	typedef hipMemcpyKind memcpykind;

	static const error success = 
		::nena::cuda::error::hipSuccess;
}};

__global__ void kernel_vector_addition(
	_In_	float1 const *a, 
	_In_	float1 const *b, 
	_Out_	float1 *c, 
	_In_	uint1 n
	)
{
	uint1 idx; idx.x = threadIdx.x;
	c[idx.x].x = a[idx.x].x + b[idx.x].x;
}


int main(int argc, char *argv [])
{
	uint1 n, ii;

	n.x = 100u;
	float1 *host_a = new float1[n.x];
	float1 *host_b = new float1[n.x];
	float1 *host_c = new float1[n.x];

	ii.x = 0;
	for (; ii.x < n.x; ii.x++)
		host_a[ii.x].x = (float) ::rand() / (float) RAND_MAX,
		host_b[ii.x].x = (float) ::rand() / (float) RAND_MAX,
		host_c[ii.x].x = (float) ::rand() / (float) RAND_MAX;

	::nena::cuda::error cs = ::nena::cuda::success;

	int deviceCount = 0;
	cs = ::hipGetDeviceCount(&deviceCount);
	if (cs != ::nena::cuda::success) 
		return EXIT_FAILURE;

	hipDeviceProp_t deviceProps;
	decltype(deviceCount) device = 0;
	for (; device < deviceCount; device++)
	{
		::hipGetDeviceProperties(
			&deviceProps, 
			device
			);
		::printf_s("cuda device %d: %d.%d\n", 
			device, 
			deviceProps.major, 
			deviceProps.minor
			);
	}

    cs = ::hipSetDevice(0);
	if (cs != ::nena::cuda::success)
		return 1;
	
	float1 *device_a = nullptr;
	float1 *device_b = nullptr;
	float1 *device_c = nullptr;

	::size_t bytes = n.x * sizeof(float1);
	auto h2d =  ::hipMemcpyKind::hipMemcpyHostToDevice;
	auto d2h =  ::hipMemcpyKind::hipMemcpyDeviceToHost;
	
	cs = ::hipMalloc((void**)&device_a, bytes);
	if (cs != ::nena::cuda::success) 
		return EXIT_FAILURE;
	cs = ::hipMalloc((void**)&device_b, bytes);
	if (cs != ::nena::cuda::success) 
		return EXIT_FAILURE;
	cs = ::hipMalloc((void**)&device_c, bytes);
	if (cs != ::nena::cuda::success) 
		return EXIT_FAILURE;

	cs = ::hipMemcpy(device_a, host_a, bytes, h2d);
	if (cs != ::nena::cuda::success) 
		return EXIT_FAILURE;
	cs = ::hipMemcpy(device_b, host_b, bytes, h2d);
	if (cs != ::nena::cuda::success) 
		return EXIT_FAILURE;


	kernel_vector_addition<<<1, n.x>>>(
		device_a, 
		device_b, 
		device_c, 
		n
		);

	cs = ::hipGetLastError();
	if (cs != ::nena::cuda::success) 
		return EXIT_FAILURE;
	cs = ::hipDeviceSynchronize();
	if (cs != ::nena::cuda::success) 
		return EXIT_FAILURE;
	cs = ::hipMemcpy(host_c, device_c, bytes, d2h);
	if (cs != ::nena::cuda::success) 
		return EXIT_FAILURE;

	hipFree(device_c);
	hipFree(device_b);
	hipFree(device_a);

    cs = hipDeviceReset();
	if (cs != ::nena::cuda::success) 
		return EXIT_FAILURE;

	ii.x = 0;
	::printf_s("<results>\n");
	for (; ii.x < n.x; ii.x++)
		::printf_s("\t [%u] \t %f + %f -> %f\n", ii.x,
		host_a[ii.x].x, host_b[ii.x].x, host_c[ii.x].x);
	::printf_s("</results>\n");
	
	delete [] host_c;
	delete [] host_b;
	delete [] host_a;

	::printf_s("press any key to exit\n");
	::getchar();
    return EXIT_SUCCESS;
}

// enable warning:
// nonstandard extension used 
#pragma warning(default: 4482)
