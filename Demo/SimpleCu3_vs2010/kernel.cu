#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <sal.h>

#include <random>

// suppress warning:
// nonstandard extension used 
#pragma warning(disable: 4482)

namespace nena { namespace cuda
{
	typedef ::hipError_t error;
	typedef ::hipMemcpyKind memcpykind;
	typedef ::hipDeviceProp_t deviceprop;

	static const error success = ::nena::cuda::error::hipSuccess;
	static const memcpykind h2d = ::nena::cuda::memcpykind::hipMemcpyHostToDevice;
	static const memcpykind d2h = ::nena::cuda::memcpykind::hipMemcpyDeviceToHost;
}};

__global__ void kernel_vector_addition(
	_In_	float1 const *a, // vector a
	_In_	float1 const *b, // vector b
	_Out_	float1 *c, // vector c
	_In_	uint1 n // vector size
	)
{
	uint1 idx; 
	idx.x = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx.x < n.x) c[idx.x].x = a[idx.x].x + b[idx.x].x;
}

int main(int argc, char *argv [])
{
	::uint1 n, ii;
	::size_t bytes;

	float1 
		*host_a = nullptr,		// vector a (cpu)
		*host_b = nullptr,		// vector b (cpu) 
		*host_c = nullptr,		// vector c (cpu)
		*device_a = nullptr,	// vector a (gpu)
		*device_b = nullptr,	// vector b (gpu)
		*device_c = nullptr;	// vector c (gpu)

	n.x = 100u; // vectors size
	bytes = n.x * sizeof(float1); // byte size

	host_a = (float1 *) ::malloc(bytes);
	host_b = (float1 *) ::malloc(bytes);
	host_c = (float1 *) ::malloc(bytes);
	::memset(host_a, 0, bytes);
	::memset(host_b, 0, bytes);
	::memset(host_c, 0, bytes);

	ii.x = 0;
	for (; ii.x < n.x; ii.x++)
		host_a[ii.x].x = (float) ::rand() / (float) RAND_MAX,
		host_b[ii.x].x = (float) ::rand() / (float) RAND_MAX,
		host_c[ii.x].x = (float) ::rand() / (float) RAND_MAX;

	::nena::cuda::error cs = ::nena::cuda::success;
	::nena::cuda::memcpykind h2d = ::nena::cuda::h2d;
	::nena::cuda::memcpykind d2h = ::nena::cuda::d2h;

	int deviceCount = 0;
	cs = ::hipGetDeviceCount(&deviceCount);
	if (cs != ::nena::cuda::success) 
		return EXIT_FAILURE;

	nena::cuda::deviceprop deviceProp;
	decltype(deviceCount) device = 0;
	for (; device < deviceCount; device++)
	{
		::hipGetDeviceProperties(
			&deviceProp, 
			device
			);
		::printf_s("cuda device %d: %d.%d\n", 
			device, 
			deviceProp.major, 
			deviceProp.minor
			);
	}

    cs = ::hipSetDevice(0);
	if (cs != ::nena::cuda::success)
		return 1;
	
	cs = ::hipMalloc((void**)&device_a, bytes);
	if (cs != ::nena::cuda::success) 
		return EXIT_FAILURE;
	cs = ::hipMalloc((void**)&device_b, bytes);
	if (cs != ::nena::cuda::success) 
		return EXIT_FAILURE;
	cs = ::hipMalloc((void**)&device_c, bytes);
	if (cs != ::nena::cuda::success) 
		return EXIT_FAILURE;

	cs = ::hipMemcpy(device_a, host_a, bytes, h2d);
	if (cs != ::nena::cuda::success) 
		return EXIT_FAILURE;
	cs = ::hipMemcpy(device_b, host_b, bytes, h2d);
	if (cs != ::nena::cuda::success) 
		return EXIT_FAILURE;

	uint3 block_size = { 1u, 1u, 1u };
	uint3 block_dims = { 1u, 1u, 1u };

	block_size.x = 16;

	block_dims.x = n.x;
	block_dims.x += block_size.x;
	block_dims.x -= 1;
	block_dims.x /= block_size.x;

	kernel_vector_addition<<<block_dims, block_size>>>(
		device_a, 
		device_b, 
		device_c, 
		n
		);

	cs = ::hipGetLastError();
	if (cs != ::nena::cuda::success) 
		return EXIT_FAILURE;
	cs = ::hipDeviceSynchronize();
	if (cs != ::nena::cuda::success) 
		return EXIT_FAILURE;
	cs = ::hipMemcpy(host_c, device_c, bytes, d2h);
	if (cs != ::nena::cuda::success) 
		return EXIT_FAILURE;

	::hipFree(device_c);
	::hipFree(device_b);
	::hipFree(device_a);

    cs = hipDeviceReset();
	if (cs != ::nena::cuda::success) 
		return EXIT_FAILURE;

	ii.x = 0;
	::printf_s("<results>\n");
	for (; ii.x < n.x; ii.x++)
		::printf_s("  [%2u] %f + %f -> %f\n", ii.x,
		host_a[ii.x].x, host_b[ii.x].x, host_c[ii.x].x);
	::printf_s("</results>\n");
	
	::free(host_c);
	::free(host_b);
	::free(host_a);

	::printf_s("press any key to exit\n");
	::getchar();
    return EXIT_SUCCESS;
}

// enable warning:
// nonstandard extension used 
#pragma warning(default: 4482)
